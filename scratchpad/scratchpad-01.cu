#include <vector>

#include "hip/hip_runtime.h"



void abort_if_cuda_error(hipError_t err)
{
    if (err != hipSuccess)
    {
        std::printf("cuda error: %s\n", hipGetErrorString(err));
        std::abort();
    }
}

void abort_if_cuda_error()
{
    hipError_t err = hipGetLastError();
    abort_if_cuda_error(err);
}

__global__ void k(double* a, std::size_t a_size)
{
    auto thread_id = threadIdx.x;
    std::printf("value: %g\n", a[thread_id]);
}

int main()
{
    std::vector<double> a(5, 20.0);
    double* d_a{};
    abort_if_cuda_error(hipMalloc((void**) &d_a, a.size() * sizeof(double)));
    abort_if_cuda_error(hipMemcpy(d_a, &a[0], a.size() * sizeof(double), hipMemcpyHostToDevice));

    k<<<1, 5>>>(d_a, a.size());
    abort_if_cuda_error();
    return 0;
}
