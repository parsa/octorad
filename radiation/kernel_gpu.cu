#include "hip/hip_runtime.h"
#include "config.hpp"
#include "kernel_gpu.hpp"

#include <array>
#include <cstdio>
#include <cstdlib>
#include <vector>

#include "hip/hip_runtime.h"
#include ""

void abort_if_cuda_error(hipError_t err)
{
    if (err != hipSuccess)
    {
        std::printf("cuda error: %s\n", hipGetErrorString(err));
        std::abort();
    }
}

void abort_if_cuda_error()
{
    hipError_t err = hipGetLastError();
    abort_if_cuda_error(err);
}

__global__ void radiation_impl(
    std::int64_t const opts_eos,
    std::int64_t const opts_problem,
    double const opts_dual_energy_sw1,
    double const opts_dual_energy_sw2,
    double const physcon_A,
    double const physcon_B,
    double const physcon_c,
    std::int64_t const er_i,
    std::int64_t const fx_i,
    std::int64_t const fy_i,
    std::int64_t const fz_i,
    std::int64_t const d,
    double const* const rho, std::size_t rho_size,
    double* const sx, std::size_t sx_size,
    double* const sy, std::size_t sy_size,
    double* const sz, std::size_t sz_size,
    double* const egas, std::size_t egas_size,
    double* const tau, std::size_t tau_size,
    double const fgamma,
    double* const U0, std::size_t U0_size,
    double* const U1, std::size_t U1_size,
    double* const U2, std::size_t U2_size,
    double* const U3, std::size_t U3_size,
    double const* const mmw, std::size_t mmw_size,
    double const* const X_spc, std::size_t X_spc_size,
    double const* const Z_spc, std::size_t Z_spc_size,
    double const dt,
    double const clightinv)
{
}

void radiation_gpu_kernel(
    std::int64_t const opts_eos,
    std::int64_t const opts_problem,
    double const opts_dual_energy_sw1,
    double const opts_dual_energy_sw2,
    double const physcon_A,
    double const physcon_B,
    double const physcon_c,
    std::int64_t const er_i,
    std::int64_t const fx_i,
    std::int64_t const fy_i,
    std::int64_t const fz_i,
    std::int64_t const d,
    std::vector<double> const& rho,
    std::vector<double>& sx,
    std::vector<double>& sy,
    std::vector<double>& sz,
    std::vector<double>& egas,
    std::vector<double>& tau,
    double const fgamma,
    std::array<std::vector<double>, NRF> U,
    std::vector<double> const mmw,
    std::vector<double> const X_spc,
    std::vector<double> const Z_spc,
    double const dt,
    double const clightinv)
{
    double* d_rho{};
    hipMalloc((void**) &d_rho, rho.size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_rho, &rho[0], rho.size(), hipMemcpyHostToDevice));
    double* d_sx{};
    hipMalloc((void**) &d_sx, sx.size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_sx, &sx[0], sx.size(), hipMemcpyHostToDevice));
    double* d_sy{};
    hipMalloc((void**) &d_sy, sy.size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_sy, &sy[0], sy.size(), hipMemcpyHostToDevice));
    double* d_sz{};
    hipMalloc((void**) &d_sz, sz.size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_sz, &sz[0], sz.size(), hipMemcpyHostToDevice));
    double* d_egas{};
    hipMalloc((void**) &d_egas, egas.size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_egas, &egas[0], egas.size(), hipMemcpyHostToDevice));
    double* d_tau{};
    hipMalloc((void**) &d_tau, tau.size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_tau, &tau[0], tau.size(), hipMemcpyHostToDevice));
    double* d_U0{};
    hipMalloc((void**) &d_U0, U[0].size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_U0, &U[0][0], U[0].size(), hipMemcpyHostToDevice));
    double* d_U1{};
    hipMalloc((void**) &d_U1, U[1].size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_U1, &U[1][0], U[1].size(), hipMemcpyHostToDevice));
    double* d_U2{};
    hipMalloc((void**) &d_U2, U[2].size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_U2, &U[2][0], U[2].size(), hipMemcpyHostToDevice));
    double* d_U3{};
    hipMalloc((void**) &d_U3, U[3].size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_U3, &U[3][0], U[3].size(), hipMemcpyHostToDevice));
    double* d_mmw{};
    hipMalloc((void**) &d_mmw, mmw.size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_mmw, &mmw[0], mmw.size(), hipMemcpyHostToDevice));
    double* d_X_spc{};
    hipMalloc((void**) &d_X_spc, X_spc.size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_X_spc, &X_spc[0], X_spc.size(), hipMemcpyHostToDevice));
    double* d_Z_spc{};
    hipMalloc((void**) &d_Z_spc, Z_spc.size() * sizeof(double));
    abort_if_cuda_error(hipMemcpy(d_Z_spc, &Z_spc[0], Z_spc.size(), hipMemcpyHostToDevice));
    radiation_impl<<<1, 1>>>(
        opts_eos,
        opts_problem,
        opts_dual_energy_sw1,
        opts_dual_energy_sw2,
        physcon_A,
        physcon_B,
        physcon_c,
        er_i,
        fx_i,
        fy_i,
        fz_i,
        d,
        d_rho, rho.size(),
        d_sx, sx.size(),
        d_sy, sy.size(),
        d_sz, sz.size(),
        d_egas, egas.size(),
        d_tau, tau.size(),
        fgamma,
        d_U0, U[0].size(),
        d_U1, U[1].size(),
        d_U2, U[2].size(),
        d_U3, U[3].size(),
        d_mmw, mmw.size(),
        d_X_spc, X_spc.size(),
        d_Z_spc, Z_spc.size(),
        dt,
        clightinv
        );
    abort_if_cuda_error();
}
